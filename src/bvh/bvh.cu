#include "hip/hip_runtime.h"
#include "bvh.h"
#include "primitive.h"
#include "../watch.h"
#include <iostream>
#include <fstream>
#include <algorithm>
#include <bitset>
#include <thrust/sort.h>
#include <thrust/execution_policy.h>

#include "../Utilities.h"
#include "../common.h"

using namespace std;
extern inline void copyFromCPUtoGPU(void** dst, void* src, int size);
extern inline void copyFromGPUtoCPU(void** dst, void* src, int size);

// Expands a 10-bit integer into 30 bits
// by inserting 2 zeros after each bit.
__device__ unsigned int d_expandBits(unsigned int v)
{
	v = (v * 0x00010001u) & 0xFF0000FFu;
	v = (v * 0x00000101u) & 0x0F00F00Fu;
	v = (v * 0x00000011u) & 0xC30C30C3u;
	v = (v * 0x00000005u) & 0x49249249u;
	return v;
}

// Calculates a 30-bit Morton code for the
// given 3D point located within the unit cube [0,1].
__device__ unsigned int d_morton3D(glm::vec3 p)
{
	float x = p.x,  y = p.y,  z = p.z;
	x = min(max(x * 1024.0f, 0.0f), 1023.0f);
	y = min(max(y * 1024.0f, 0.0f), 1023.0f);
	z = min(max(z * 1024.0f, 0.0f), 1023.0f);
	unsigned int xx = d_expandBits((unsigned int)x);
	unsigned int yy = d_expandBits((unsigned int)y);
	unsigned int zz = d_expandBits((unsigned int)z);
	return xx * 4 + yy * 2 + zz;
}


__global__ void get_bb(int num, int m, Primitive* d_primitives, BBox* d_bb)
{
	unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= num +1)
		return;
	int div = m / num;
	int res = m%num;
	if (index == num + 1)
	{
		BBox tem_bbox;
		for (int i = m - res; i < m; i++)
		{
			tem_bbox.expand(d_primitives[i].d_get_expand_bbox());
		}
		d_bb[index] = tem_bbox;
	}
	else
	{
		BBox tem_bbox;
		for (int i = 0; i < div; i++)  //use shared to replace
		{
			tem_bbox.expand(d_primitives[i*num + index].d_get_expand_bbox());
		}
		d_bb[index].expand(tem_bbox);
	}
}

__global__ void compute_morton_bbox(int num, Primitive* d_primitives, BBox bb, MortonCode* mortons, BBox* bboxes)
{
	unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= num)
		return;
	BBox tem_bbox = d_primitives[index].d_get_expand_bbox();
	bboxes[index] = tem_bbox;
	mortons[index] = d_morton3D(bb.getUnitcubePosOf(tem_bbox.centroid()));
}

// Expands a 10-bit integer into 30 bits
// by inserting 2 zeros after each bit.
unsigned int BVHAccel::expandBits(unsigned int v)
{
	v = (v * 0x00010001u) & 0xFF0000FFu;
	v = (v * 0x00000101u) & 0x0F00F00Fu;
	v = (v * 0x00000011u) & 0xC30C30C3u;
	v = (v * 0x00000005u) & 0x49249249u;
	return v;
}

// Calculates a 30-bit Morton code for the
// given 3D point located within the unit cube [0,1].
unsigned int BVHAccel::morton3D(float x, float y, float z)
{
	x = min(max(x * 1024.0f, 0.0f), 1023.0f);
	y = min(max(y * 1024.0f, 0.0f), 1023.0f);
	z = min(max(z * 1024.0f, 0.0f), 1023.0f);
	unsigned int xx = expandBits((unsigned int)x);
	unsigned int yy = expandBits((unsigned int)y);
	unsigned int zz = expandBits((unsigned int)z);
	return xx * 4 + yy * 2 + zz;
}

/**
* a wrapper to calculate morton code from
* the position of an object inside the
* unit cube.
*/
unsigned int BVHAccel::morton3D(glm::vec3 pos)
{
	return morton3D(pos.x, pos.y, pos.z);
}

/**
* comparer used to sort primitives acoording
* to their morton code.
*/


BBox BVHAccel::computet_root_bbox(Primitive* d_tem_primitives)
{
	const unsigned int num_threads = 128;
	vector<BBox> c_bb(num_threads + 1);
	BBox* d_bb;
	
	copyFromCPUtoGPU((void**)&d_bb, &c_bb[0], sizeof(BBox)* c_bb.size());
	get_bb << <1, c_bb.size() >> > (num_threads, _primitives.size(), d_tem_primitives, d_bb);

	BBox* cc_bb, bb;
	copyFromGPUtoCPU((void**)&cc_bb, d_bb, sizeof(BBox)*c_bb.size());
	for (int i = 0; i < c_bb.size(); i++)
	{
		bb.expand(cc_bb[i]);
	}

	hipFree(d_bb);

	return bb;
}

void save(vector<Primitive>& primitives, string file_name)
{
	//ofstream outfile(file_name);
	//outfile << "# morton code" << endl;
	//for (auto pri: primitives)
	//{
	//	outfile << pri.morton_code << endl;   //����д���ļ�
	//}
	//outfile.close();
	//cout << "save done!" << endl;
}

void BVHAccel::compute_bbox_and_morton()
{
	Primitive* d_tem_primitives;
	MortonCode* d_tem_morton_codes;
	BBox* d_tem_bboxes;
	_morton_codes.resize(_primitives.size());
	_bboxes.resize(_primitives.size());

	copyFromCPUtoGPU((void**)&d_tem_primitives, &_primitives[0], sizeof(Primitive)*_primitives.size());
	copyFromCPUtoGPU((void**)&d_tem_morton_codes, &_morton_codes[0], sizeof(MortonCode)*_morton_codes.size());
	copyFromCPUtoGPU((void**)&d_tem_bboxes, &_bboxes[0], sizeof(BBox)*_bboxes.size());

	BBox bb = computet_root_bbox(d_tem_primitives);

	unsigned int numThreads, numBlocks;
	unsigned int blockSize = 512;
	unsigned int n = _primitives.size();
	numThreads = min(blockSize, n);
	numBlocks = (n % numThreads != 0) ? (n / numThreads + 1) : (n / numThreads);

	compute_morton_bbox << <numBlocks, numThreads >> > (n, d_tem_primitives, bb, d_tem_morton_codes, d_tem_bboxes);

	hipMemcpy(&_morton_codes[0], d_tem_morton_codes, sizeof(MortonCode)*_morton_codes.size(), hipMemcpyDeviceToHost);
	hipMemcpy(&_bboxes[0], d_tem_bboxes, sizeof(BBox)*_bboxes.size(), hipMemcpyDeviceToHost);

	hipFree(d_tem_primitives);
	hipFree(d_tem_morton_codes);
	hipFree(d_tem_bboxes);
}

__global__ void init_nodes(BRTreeNode* _nodes,const unsigned int num)
{
	unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= num)
		return;

	BRTreeNode node;
	node.setIdx(index);
	node.bbox = BBox();

	_nodes[index] = node;
}

void BVHAccel::init()
{
	d_bvh = new D_BVH();

	auto size = _sorted_primitives.size();
	numInternalNode = size - 1;
	numLeafNode = size;

	//whether to set h_vertices = NULL before send to gpu?
	copyFromCPUtoGPU((void**)&d_bvh->d_primitives, &_sorted_primitives[0], sizeof(Primitive)*_sorted_primitives.size());
	copyFromCPUtoGPU((void**)&d_sorted_morton_code, &_sorted_morton_codes[0], sizeof(MortonCode)*_sorted_morton_codes.size());
	copyFromCPUtoGPU((void**)&d_bboxes, &_sorted_bboxes[0], sizeof(BBox)*_sorted_bboxes.size());

	//initialize d_leaf_nodes and d_internal_nodes: with a parallel way? ?????
	hipMalloc((void**)&d_bvh->d_leaf_nodes, numLeafNode * sizeof(BRTreeNode));
	hipMalloc((void**)&d_bvh->d_internal_nodes, numInternalNode * sizeof(BRTreeNode));

	int threadPerBlock = DEFAULT_THREAD_PER_BLOCK;
	int numBlock = (numLeafNode + DEFAULT_THREAD_PER_BLOCK - 1) / threadPerBlock;
	init_nodes << <numBlock, threadPerBlock >> > (d_bvh->d_leaf_nodes, numLeafNode);


	threadPerBlock = DEFAULT_THREAD_PER_BLOCK;
	numBlock = (numInternalNode + DEFAULT_THREAD_PER_BLOCK - 1) / threadPerBlock;
	init_nodes << <numBlock, threadPerBlock >> > (d_bvh->d_internal_nodes, numInternalNode);
}

void BVHAccel::build()
{
	//build the bvh
	int threadPerBlock = DEFAULT_THREAD_PER_BLOCK;
	int numBlock = (numInternalNode + DEFAULT_THREAD_PER_BLOCK - 1) / threadPerBlock;
	processInternalNode << <numBlock, threadPerBlock >> > (d_sorted_morton_code, numInternalNode,
		d_bvh->d_leaf_nodes, d_bvh->d_internal_nodes);

	//calculate bounding box
	threadPerBlock = DEFAULT_THREAD_PER_BLOCK;
	numBlock = (numLeafNode + DEFAULT_THREAD_PER_BLOCK - 1) / threadPerBlock;
	calculateBoudingBox << <numBlock, threadPerBlock >> > (d_bboxes, numLeafNode,
		d_bvh->d_leaf_nodes, d_bvh->d_internal_nodes);
}

void BVHAccel::init_primitives(Mesh& body)
{
	//prepare primitives
	obj_vertices.resize(body.vertices.size());
	for (int i = 0; i < body.vertices.size(); i++)
	{
		obj_vertices[i] = glm::vec3(body.vertices[i]);
	}

	safe_cuda(hipMalloc((void**)&d_obj_vertices, sizeof(glm::vec3)*obj_vertices.size()));
	safe_cuda(hipMemcpy(d_obj_vertices, &obj_vertices[0], sizeof(glm::vec3)*obj_vertices.size(), hipMemcpyHostToDevice));

	//create primitives
	glm::vec3* h_obj_vertices = &obj_vertices[0];
	_primitives.resize(body.vertex_indices.size() / 3);

	for (int i = 0; i < _primitives.size(); i++)
	{
		Primitive tem_pri(h_obj_vertices, d_obj_vertices, body.vertex_indices[i * 3 + 0],
			body.vertex_indices[i * 3 + 1],
			body.vertex_indices[i * 3 + 2]);
		_primitives[i] = tem_pri;
	}
}

BVHAccel::BVHAccel(Mesh& body, size_t max_leaf_size):

	d_bboxes(nullptr),
#ifdef _DEBUG
	h_leaf_nodes(nullptr),
	h_internal_nodes(nullptr),
#endif
	d_sorted_morton_code(nullptr)
{
	init_primitives(body);

	// edge case
	if (_primitives.empty()) {
		return;
	}

	compute_bbox_and_morton();


	// remove duplicates
	vector<unsigned int> indices;
	indices_sort(_morton_codes, indices);
	remove_redundant(_morton_codes, indices);

	filter(_morton_codes, indices, _sorted_morton_codes);
	filter(_primitives, indices, _sorted_primitives);
	filter(_bboxes, indices, _sorted_bboxes);

	// init	GPU data, including d_bboxes,d_primitives, d_sorted_morton_code,d_leaf_nodes, d_internal_nodes 
	init();

	// build the brt tree
	build();
}

BVHAccel::~BVHAccel()
{
	hipFree(d_bboxes);
	hipFree(d_sorted_morton_code);
	hipFree(d_obj_vertices);

	// Free d_bvh here cause it has pointer points to gpu memory
	// and we need to pass the value several times and make sure the 
	// resource not freed, so we can't free it in its own destructor. 
	// \BVHAccel controls the lifetieme of \d_bvh, if the destructor 
	// of \BVHAccel called, which means we can free all the resources in 
	// gpu and cpu(Obviously, this violates the "new" and "free" pair priciple)

	d_bvh->free_memory();
}

#ifdef _DEBUG
BRTreeNode* BVHAccel::get_leaf_nodes()
{
	copyFromGPUtoCPU((void**)&h_leaf_nodes, d_bvh->d_leaf_nodes, numLeafNode * sizeof(BRTreeNode));
	return h_leaf_nodes;
}
BRTreeNode* BVHAccel::get_internal_nodes()
{
	copyFromGPUtoCPU((void**)&h_internal_nodes, d_bvh->d_internal_nodes, numInternalNode * sizeof(BRTreeNode));
	return h_internal_nodes;
}
BRTreeNode* BVHAccel::get_root() const
{
	return &h_internal_nodes[0];
}
BRTreeNode* BVHAccel::get_left_child(BRTreeNode* node)const
{
	bool is_leaf = false;
	bool is_null = false;
	int  child_idx = false;
	child_idx = node->getChildA(is_leaf, is_null);
	if (!is_null)
	{
		if (is_leaf)
		{
			return &h_leaf_nodes[child_idx];
		}
		else
		{
			return &h_internal_nodes[child_idx];
		}
	}
	else
		return nullptr;
}
BRTreeNode* BVHAccel::get_right_child(BRTreeNode* node)const
{
	bool is_leaf = false;
	bool is_null = false;
	int  child_idx = false;
	child_idx = node->getChildB(is_leaf, is_null);
	if (!is_null)
	{
		if (is_leaf)
		{
			return &h_leaf_nodes[child_idx];
		}
		else
		{
			return &h_internal_nodes[child_idx];
		}
	}
	else
		return nullptr;
}
bool BVHAccel::is_leaf(BRTreeNode* node)const
{
	bool is_leaf = false;
	bool is_null_a = false;
	bool is_null_b = false;
	int  child_idx_a = false;
	int  child_idx_b = false;
	child_idx_a = node->getChildA(is_leaf, is_null_a);
	child_idx_b = node->getChildB(is_leaf, is_null_b);

	if (is_null_a && is_null_b)
		return true;
	return false;

}
bool BVHAccel::intersect(const glm::vec3 point, int& idx) const
{
	// Allocate traversal stack from thread-local memory,
	// and push NULL to indicate that there are no postponed nodes.
	BRTreeNode* stack[64];
	BRTreeNode** stackPtr = stack;
	*stackPtr++ = NULL; // push

						// Traverse nodes starting from the root.
	BRTreeNode* node = get_root();
	do
	{
		// Check each child node for overlap.
		BRTreeNode* childA = get_left_child(node);
		BRTreeNode* childB = get_right_child(node);
		bool overlapL = check_overlap(point, childA);
		bool overlapR = check_overlap(point, childB);

		// Query overlaps a leaf node => report collision with the first collision.
		if (overlapL && is_leaf(childA))
		{
			idx = childA->getIdx();
			//idx = -(idx + 1);   //is a leaf, and we can get it through primitive[idx]
			return true;
		}

		if (overlapR && is_leaf(childB))
		{
			idx = childB->getIdx();
			//idx = -(idx + 1);   //is a leaf
			return true;
		}

		// Query overlaps an internal node => traverse.
		bool traverseL = (overlapL && !is_leaf(childA));
		bool traverseR = (overlapR && !is_leaf(childB));

		if (!traverseL && !traverseR)
			node = *--stackPtr; // pop
		else
		{
			node = (traverseL) ? childA : childB;
			if (traverseL && traverseR)
				*stackPtr++ = childB; // push
		}
	} while (node != NULL);
	return false;
}
bool BVHAccel::check_overlap(const glm::vec3 point, BRTreeNode* node)const
{
	return node->bbox.intersect(point);
}
void BVHAccel::access(BRTreeNode* root, vector<BRTreeNode*>& bad_bode)
{
	if (root->bbox.min.x > root->bbox.max.x)
	{
		if (is_leaf(root))
		{
			bad_bode.push_back(root);
			return;
		}
		else
		{
			access(get_left_child(root), bad_bode);
			access(get_right_child(root), bad_bode);
		}
	}


}
void BVHAccel::copy_data_gpu_to_cpu()
{
	copyFromGPUtoCPU((void**)&h_internal_nodes, d_bvh->d_internal_nodes, sizeof(BRTreeNode)*numInternalNode);
	copyFromGPUtoCPU((void**)&h_leaf_nodes, d_bvh->d_leaf_nodes, sizeof(BRTreeNode)*numLeafNode);

}

// call copy_data_gpu_to_cpu() before print
void BVHAccel::print(BRTreeNode* root, int depth, const int max_depth)
{
	depth++;
	if (depth > max_depth)
		return;
	bool is_null = false;
	cout << root->getIdx() << " " << root->getParent(is_null);
	root->bbox.print();

	if (is_leaf(root))
	{
		return;
	}
	else
	{
		is_null = false;
		cout << " left:" << get_left_child(root)->getIdx() << " " << get_left_child(root)->getParent(is_null);  get_left_child(root)->bbox.print();
		is_null = false;
		cout << " right:" << get_right_child(root)->getIdx() << " "<<  get_right_child(root)->getParent(is_null);  get_right_child(root)->bbox.print();

		print(get_left_child(root),depth +1, max_depth);
		print(get_right_child(root), depth + 1, max_depth);
	}
}

// call copy_data_gpu_to_cpu() before draw
void BVHAccel::draw(BRTreeNode* root)
{
	//root->bbox.draw();
	bool is_null = false;
	cout << root->getIdx() << " parent_id: " << root->getParent(is_null) << "  ";

	bool is_leaf_a = false;
	bool is_null_a = false;
	bool is_null_b = false;
	int  child_idx_a = false;
	int  child_idx_b = false;
	child_idx_a = root->getChildA(is_leaf_a, is_null_a);
	cout << "left_id " << child_idx_a << " is_leaf_a" << is_leaf_a;

	child_idx_b = root->getChildB(is_leaf_a, is_null_b);
	cout << "right_id " << child_idx_b << " is_leaf_a" << is_leaf_a;
	root->bbox.print();

	if (is_leaf(root))
	{
		//cout << "is_leaf";
		//bool is_leaf = false;
		//bool is_null_a = false;
		//bool is_null_b = false;
		//int  child_idx_a = false;
		//int  child_idx_b = false;
		//child_idx_a = root->getChildA(is_leaf, is_null_a);
		//cout << "left_id " << child_idx_a << " is_leaf" << is_leaf;

		//child_idx_b = root->getChildB(is_leaf, is_null_b);
		//cout << "right_id " << child_idx_b << " is_leaf" << is_leaf;

		return;
	}
	else
	{
		draw(get_left_child(root));
		draw(get_right_child(root));
	}
}
#endif



