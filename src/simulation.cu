#include "hip/hip_runtime.h"

#include "simulator.h"
#include "spring.h"
#include "Mesh.h"
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include <iostream>
#include <fstream>

#include "watch.h"
#include "common.h"

using namespace std;

__global__ void compute_face_normal(glm::vec3* g_pos_in, unsigned int* cloth_index, const unsigned int cloth_index_size, glm::vec3* cloth_face);   //update cloth face normal
__global__ void verlet(glm::vec3 * g_pos_in, glm::vec3 * g_pos_old_in, glm::vec3 * g_pos_out, glm::vec3 * g_pos_old_out,
						unsigned int* CSR_R_STR, s_spring* CSR_C_STR, unsigned int* CSR_R_BD, s_spring* CSR_C_BD,
						D_BVH bvh, glm::vec3* d_collision_force,
						const unsigned int NUM_VERTICES);  //verlet intergration
__global__ void update_vbo_pos(glm::vec4* pos_vbo, glm::vec3* pos_cur, const unsigned int NUM_VERTICES);
__global__ void compute_vbo_normal(glm::vec3* normals, unsigned int* CSR_R, unsigned int* CSR_C_adjface_to_vertex, glm::vec3* face_normal, const unsigned int NUM_VERTICES);

Simulator::Simulator()
{
	
}

Simulator::~Simulator()
{
	hipFree(x_cur[0]);
	hipFree(x_cur[1]);
	hipFree(x_last[0]);
	hipFree(x_last[1]);
	hipFree(d_collision_force);
	hipFree(d_CSR_R);
	hipFree(d_CSR_C_adjface_to_vertex);
	hipFree(d_face_normals);

	hipFree(CSR_R_structure);
	hipFree(CSR_R_bend);
	hipFree(CSR_C_structure);
	hipFree(CSR_C_bend);
	for(auto& c : cuda_bvh)
		delete c;
}

Simulator::Simulator(Mesh& sim_cloth, std::vector<Mesh>& body) :readID(0), writeID(1)
{
	init_cloth(sim_cloth);
	init_spring(sim_cloth);
	build_bvh(body);
}

void Simulator::init_cloth(Mesh& sim_cloth)
{
	// \d_vbo_array_resource points to cloth's array buffer  
	safe_cuda(hipGraphicsGLRegisterBuffer(&d_vbo_array_resource, sim_cloth.vbo.array_buffer, cudaGraphicsMapFlagsWriteDiscard));   	//register vbo


	//set heap size, the default is 8M
	size_t heap_size = 256 * 1024 * 1024;  
	hipDeviceSetLimit(hipLimitMallocHeapSize, heap_size);

	// Send the cloth's vertices to GPU
	const unsigned int vertices_bytes = sizeof(glm::vec3) * sim_cloth.vertices.size();
	safe_cuda(hipMalloc((void**)&x_cur[0], vertices_bytes));			 // cloth vertices
	safe_cuda(hipMalloc((void**)&x_cur[1], vertices_bytes));			 // cloth vertices
	safe_cuda(hipMalloc((void**)&x_last[0], vertices_bytes));	 // cloth old vertices
	safe_cuda(hipMalloc((void**)&x_last[1], vertices_bytes));	 // cloth old vertices
	safe_cuda(hipMalloc((void**)&d_collision_force, sizeof(glm::vec3) * sim_cloth.vertices.size()));  //collision response force
	safe_cuda(hipMemset(d_collision_force, 0, sizeof(glm::vec3) * sim_cloth.vertices.size()));    //initilize to 0

	x_cur_in = x_cur[readID];
	x_cur_out = x_cur[writeID];
	x_last_in = x_last[readID];
	x_last_out = x_last[writeID];

	vector<glm::vec3> tem_vertices(sim_cloth.vertices.size());
	for (int i=0;i< sim_cloth.vertices.size();i++)
	{
		tem_vertices[i] = glm::vec3(sim_cloth.vertices[i]);   // glm::vec4 -> glm::vec3
	}

	safe_cuda(hipMemcpy(x_cur[0], &tem_vertices[0], vertices_bytes, hipMemcpyHostToDevice));
	safe_cuda(hipMemcpy(x_last[0], &tem_vertices[0], vertices_bytes, hipMemcpyHostToDevice));

	//����normal��������ݣ�ÿ�����ڽӵ�������� + ÿ�����3���������
	vector<unsigned int> TEM_CSR_R;
	vector<unsigned int> TEM_CSR_C_adjface;
	get_vertex_adjface(sim_cloth, TEM_CSR_R, TEM_CSR_C_adjface);

	safe_cuda(hipMalloc((void**)&d_CSR_R, sizeof(unsigned int) * TEM_CSR_R.size()));
	safe_cuda(hipMalloc((void**)&d_CSR_C_adjface_to_vertex, sizeof(unsigned int) * TEM_CSR_C_adjface.size()));
	safe_cuda(hipMemcpy(d_CSR_R, &TEM_CSR_R[0], sizeof(unsigned int) * TEM_CSR_R.size(), hipMemcpyHostToDevice));
	safe_cuda(hipMemcpy(d_CSR_C_adjface_to_vertex, &TEM_CSR_C_adjface[0], sizeof(unsigned int) * TEM_CSR_C_adjface.size(), hipMemcpyHostToDevice));
	
	safe_cuda(hipMalloc((void**)&d_face_normals, sizeof(glm::vec3) * sim_cloth.faces.size()));    //face normal

	safe_cuda(hipGraphicsGLRegisterBuffer(&d_vbo_index_resource, sim_cloth.vbo.index_buffer, cudaGraphicsMapFlagsWriteDiscard));   	//register vbo
}

void Simulator::init_spring(Mesh& sim_cloth)
{
	cout << "build springs" << endl;
	// Construct structure and bend springs in GPU
	Springs springs(&sim_cloth);
	
	vector<unsigned int> TEM_CSR_R_structure, TEM_CSR_R_bend;
	vector<s_spring> TEM_CSR_C_structure, TEM_CSR_C_bend;

	springs.CSR_structure_spring(&sim_cloth, TEM_CSR_R_structure, TEM_CSR_C_structure);
	springs.CSR_bend_spring(&sim_cloth, TEM_CSR_R_bend, TEM_CSR_C_bend);

	safe_cuda(hipMalloc((void**)&CSR_R_structure, TEM_CSR_R_structure.size() * sizeof(unsigned int)));
	safe_cuda(hipMalloc((void**)&CSR_R_bend, TEM_CSR_R_bend.size() * sizeof(unsigned int)));
	safe_cuda(hipMalloc((void**)&CSR_C_structure, TEM_CSR_C_structure.size() * sizeof(s_spring)));
	safe_cuda(hipMalloc((void**)&CSR_C_bend, TEM_CSR_C_bend.size() * sizeof(s_spring)));

	safe_cuda(hipMemcpy(CSR_R_structure, &TEM_CSR_R_structure[0], TEM_CSR_R_structure.size() * sizeof(unsigned int), hipMemcpyHostToDevice));
	safe_cuda(hipMemcpy(CSR_R_bend, &TEM_CSR_R_bend[0], TEM_CSR_R_bend.size() * sizeof(unsigned int), hipMemcpyHostToDevice));
	safe_cuda(hipMemcpy(CSR_C_structure, &TEM_CSR_C_structure[0], TEM_CSR_C_structure.size() * sizeof(s_spring), hipMemcpyHostToDevice));
	safe_cuda(hipMemcpy(CSR_C_bend, &TEM_CSR_C_bend[0], TEM_CSR_C_bend.size() * sizeof(s_spring), hipMemcpyHostToDevice));
	
	cout << "springs build successfully!" << endl;
}

void Simulator::build_bvh(std::vector<Mesh>& bodyvec)
{
	stop_watch watch;
	watch.start();
	for(auto & body : bodyvec) {
		Mesh bvh_body = body;   // for bvh consttruction
		bvh_body.vertex_extend(0.003);

		watch.start();
		cuda_bvh.emplace_back(new BVHAccel(bvh_body));
		watch.stop();
		cout << "bvh build done free time elapsed: " << watch.elapsed() << "us" << endl;
	}
}


void Simulator::simulate(Mesh* sim_cloth)
{
	//cuda kernel compute .........
	static int i = 0;
	static int substeps = 10;
	cuda_verlet((i++ % (cuda_bvh.size() * substeps)) / substeps, sim_cloth->vertices.size());

	cuda_update_vbo(sim_cloth);     // update array buffer for opengl

	swap_buffer();
}

void Simulator::get_vertex_adjface(Mesh& sim_cloth, vector<unsigned int>& CSR_R, vector<unsigned int>& CSR_C_adjface)
{
	vector<vector<unsigned int>> adjaceny(sim_cloth.vertices.size());
	for(int i=0;i<sim_cloth.faces.size();i++)
	{
		unsigned int f[3];
		for(int j=0;j<3;j++)
		{
			f[j] = sim_cloth.faces[i].vertex_index[j];
			adjaceny[f[j]].push_back(i);
		}
	}

	// i-th vertex adjacent face start_index = CSR_R[i], end_index = CSR_R[i+1]
	// then you can acess CSR_C_adjface[start_index->end_index]
	unsigned int start_idx = 0;
	for(int i=0;i<adjaceny.size();i++)
	{
		CSR_R.push_back(start_idx);
		start_idx += adjaceny[i].size();

		for(int j=0;j<adjaceny[i].size();j++)
		{
			CSR_C_adjface.push_back(adjaceny[i][j]);
		}
	}

	CSR_R.push_back(start_idx);
}

void Simulator::cuda_verlet(int frameidx, const unsigned int numParticles)
{
	unsigned int numThreads, numBlocks;
	
	computeGridSize(numParticles, 512, numBlocks, numThreads);
	verlet <<< numBlocks, numThreads >>>(x_cur_in,x_last_in, x_cur_out, x_last_out,
										CSR_R_structure, CSR_C_structure, CSR_R_bend, CSR_C_bend,
										*(cuda_bvh[frameidx]->d_bvh), d_collision_force,
										numParticles);

	// stop the CPU until the kernel has been executed
	safe_cuda(hipDeviceSynchronize());
}

void Simulator::cuda_update_vbo(Mesh* sim_cloth)
{
	unsigned int numParticles = sim_cloth->vertices.size();

	size_t num_bytes;
	glm::vec4* d_vbo_vertex;           //point to vertex address in the OPENGL buffer
	glm::vec3* d_vbo_normal;           //point to normal address in the OPENGL buffer
	unsigned int* d_adjvertex_to_face;    // the order like this: f0(v0,v1,v2) -> f1(v0,v1,v2) -> ... ->fn(v0,v1,v2)
	
	safe_cuda(hipGraphicsMapResources(1, &d_vbo_array_resource));
	safe_cuda(hipGraphicsMapResources(1, &d_vbo_index_resource));
	safe_cuda(hipGraphicsResourceGetMappedPointer((void **)&d_vbo_vertex, &num_bytes, d_vbo_array_resource));
	safe_cuda(hipGraphicsResourceGetMappedPointer((void **)&d_adjvertex_to_face, &num_bytes, d_vbo_index_resource));

	d_vbo_normal = (glm::vec3*)((float*)d_vbo_vertex + 4 * sim_cloth->vertices.size() + 2 * sim_cloth->tex.size());   // ��ȡnormalλ��ָ��	

	unsigned int numThreads, numBlocks;

	// update vertex position
	computeGridSize(numParticles, 512, numBlocks, numThreads);
	update_vbo_pos << < numBlocks, numThreads >> > (d_vbo_vertex, x_cur_out, numParticles);
	safe_cuda(hipDeviceSynchronize());  	// stop the CPU until the kernel has been executed

	// we need to compute face normal before computing vbo normal
	computeGridSize(sim_cloth->faces.size(), 512, numBlocks, numThreads);
	compute_face_normal << <numBlocks, numThreads >> > (x_cur_in, d_adjvertex_to_face, sim_cloth->vertex_indices.size(), d_face_normals);
	safe_cuda(hipDeviceSynchronize());

	// update vertex normal
	computeGridSize(numParticles, 1024, numBlocks, numThreads);
	compute_vbo_normal << < numBlocks, numThreads >> > (d_vbo_normal, d_CSR_R, d_CSR_C_adjface_to_vertex, d_face_normals ,numParticles);
	safe_cuda(hipDeviceSynchronize());

	safe_cuda(hipGraphicsUnmapResources(1, &d_vbo_index_resource));
	safe_cuda(hipGraphicsUnmapResources(1, &d_vbo_array_resource));
}

void Simulator::save(string file_name)
{
}

void Simulator::computeGridSize(unsigned int n, unsigned int blockSize, unsigned int &numBlocks, unsigned int &numThreads)
{
	numThreads = min(blockSize, n);
	numBlocks = (n % numThreads != 0) ? (n / numThreads + 1) : (n / numThreads);
}

void Simulator::swap_buffer()
{
	swap(readID, writeID);

	x_cur_in = x_cur[readID];
	x_cur_out = x_cur[writeID];
	x_last_in = x_last[readID];
	x_last_out = x_last[writeID];
}

void Simulator::update_vertex(glm::vec3 new_value, const unsigned int idx)
{
	safe_cuda(hipMemcpy(&x_cur_in[idx], &new_value[0], sizeof(glm::vec3), hipMemcpyHostToDevice));
	safe_cuda(hipMemcpy(&x_last_in[idx], &new_value[0], sizeof(glm::vec3), hipMemcpyHostToDevice));
}

